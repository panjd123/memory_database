#include "utils.cuh"
#include "SSB.cuh"
#include "TPCH.cuh"
#include <hip/hip_runtime.h>

int main(int argc, char** argv) {
    // int device;
    // hipGetDevice(&device);
    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, device);
    // std::cout << "SM count: " << prop.multiProcessorCount << std::endl;
    {
        using namespace TPCH;
        auto args = generate(argc, argv);
        benchmark(args);
    }
}