#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK(call) \
  if ((call) != hipSuccess) { \
    fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(call)); \
    exit(EXIT_FAILURE); \
  }

__device__ unsigned int lcg_random(unsigned int* state) {
  const unsigned int a = 1664525u;
  const unsigned int c = 1013904223u;
  *state = a * (*state) + c;
  return *state;
}

__global__ void seq_read_float4_kernel(const float4* __restrict__ d_data, int N, float* __restrict__ d_sink) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  float val = 0.f;
  for (int i = tid; i < N; i += stride) {
    float4 v = d_data[i];
    val += v.x + v.y + v.z + v.w;
  }
  if (tid == 0) d_sink[0] = val;
}

__global__ void seq_read_float_kernel(const float* __restrict__ d_data, int N, float* __restrict__ d_sink) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  float val = 0.f;
  for (int i = tid; i < N; i += stride) {
    val += d_data[i];
  }
  if (tid == 0) d_sink[0] = val;
}

__global__ void rand_read_float_lcg_kernel(const float* __restrict__ d_data, int N, float* __restrict__ d_sink) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  unsigned int state = tid + 12345;
  float val = 0.f;
  for (int i = tid; i < N; i += stride) {
    unsigned int idx = lcg_random(&state) % N;
    val += d_data[idx];
  }
  if (tid == 0) d_sink[0] = val;
}

__global__ void rand_read_float4_lcg_kernel(const float4* __restrict__ d_data, int N, float* __restrict__ d_sink) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  unsigned int state = tid + 12345;
  float val = 0.f;
  for (int i = tid; i < N; i += stride) {
    unsigned int idx = lcg_random(&state) % N;
    float4 v = d_data[idx];
    val += v.x + v.y + v.z + v.w;
  }
  if (tid == 0) d_sink[0] = val;
}

int main() {
  const int threads = 256;
  const int blocks = 256;

  const int test_sizes_mb[] = {4, 8, 16, 32, 64, 128, 256, 512};
  const int num_tests = sizeof(test_sizes_mb) / sizeof(test_sizes_mb[0]);

  float4* d_data_float4 = nullptr;
  float* d_data_float = nullptr;
  float* d_sink = nullptr;

  CHECK(hipMalloc(&d_sink, sizeof(float)));

  printf("DataSizeMB,TestType,BandwidthGBps\n");

  for (int test_i = 0; test_i < num_tests; ++test_i) {
    int mb = test_sizes_mb[test_i];
    size_t bytes_float = (size_t)mb * 1024 * 1024;
    int num_float = (int)(bytes_float / sizeof(float));
    int num_float4 = num_float / 4;

    if (d_data_float4) hipFree(d_data_float4);
    if (d_data_float) hipFree(d_data_float);

    CHECK(hipMalloc(&d_data_float4, sizeof(float4) * num_float4));
    CHECK(hipMalloc(&d_data_float, sizeof(float) * num_float));

    CHECK(hipMemset(d_data_float4, 0, sizeof(float4) * num_float4));
    CHECK(hipMemset(d_data_float, 0, sizeof(float) * num_float));
    CHECK(hipMemset(d_sink, 0, sizeof(float)));

    seq_read_float4_kernel<<<blocks, threads>>>(d_data_float4, num_float4, d_sink);
    seq_read_float_kernel<<<blocks, threads>>>(d_data_float, num_float, d_sink);
    rand_read_float_lcg_kernel<<<blocks, threads>>>(d_data_float, num_float, d_sink);
    CHECK(hipDeviceSynchronize());

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    float ms;

    // seq_float4
    CHECK(hipEventRecord(start));
    seq_read_float4_kernel<<<blocks, threads>>>(d_data_float4, num_float4, d_sink);
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&ms, start, stop));
    float gbps_seq_float4 = (bytes_float / (1024.f*1024.f*1024.f)) / (ms / 1000.f);
    printf("%d,seq_float4,%.3f\n", mb, gbps_seq_float4);

    // seq_float
    CHECK(hipEventRecord(start));
    seq_read_float_kernel<<<blocks, threads>>>(d_data_float, num_float, d_sink);
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&ms, start, stop));
    float gbps_seq_float = (bytes_float / (1024.f*1024.f*1024.f)) / (ms / 1000.f);
    printf("%d,seq_float,%.3f\n", mb, gbps_seq_float);

    // rand_float4
    CHECK(hipEventRecord(start));
    rand_read_float4_lcg_kernel<<<blocks, threads>>>(d_data_float4, num_float4, d_sink);
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&ms, start, stop));
    float gbps_rand_float4 = (bytes_float / (1024.f*1024.f*1024.f)) / (ms / 1000.f);
    printf("%d,rand_float4,%.3f\n", mb, gbps_rand_float4);

    // rand_float
    CHECK(hipEventRecord(start));
    rand_read_float_lcg_kernel<<<blocks, threads>>>(d_data_float, num_float, d_sink);
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&ms, start, stop));
    float gbps_rand_float = (bytes_float / (1024.f*1024.f*1024.f)) / (ms / 1000.f);
    printf("%d,rand_float,%.3f\n", mb, gbps_rand_float);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
  }

  hipFree(d_data_float4);
  hipFree(d_data_float);
  hipFree(d_sink);

  return 0;
}
